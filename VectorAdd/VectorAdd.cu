#include <iostream>
#include <hip/hip_runtime.h>

__global__ void GPUAdd( int *a, int *b, int *c, int no_elements)
{
        c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

int main()
{
    int no_elements =32;
    int a_host[no_elements];
    int b_host[no_elements];
    int c_host[no_elements];
    
    for(int i = 0; i < no_elements; i++)
    {
        a_host[i] = i;
        b_host[i] = 12*i;
    }

    int *a_dev, *b_dev, *c_dev;
    hipMalloc(&a_dev, no_elements*sizeof(int));
    hipMalloc(&b_dev, no_elements*sizeof(int));
    hipMalloc(&c_dev, no_elements*sizeof(int));
    
    hipMemcpy(a_dev, a_host, no_elements*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(b_dev, b_host, no_elements*sizeof(int), hipMemcpyHostToDevice);

    GPUAdd<<<1,no_elements>>>(a_dev, b_dev, c_dev, no_elements);
    hipMemcpy(c_host, c_dev, no_elements*sizeof(int), hipMemcpyDeviceToHost);
    
    hipFree(a_dev);
    hipFree(b_dev);
    hipFree(c_dev);
    
    for(int i = 0; i < no_elements; i++)
        std::cout << "GPU computed resutl: " << c_host[i] << std::endl;

    return 0;

}


