#include "hip/hip_runtime.h"
#include<gpu.h>

__global__ void GADD(int *a , int *b, int *c)
{
    c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}


void AddGPU( int *a, int *b, int *c, int no_elem)
{
    std::cout << "FROM GPU" << std::endl;
    int *a_dev,*b_dev,*c_dev;
    hipMalloc(&a_dev, no_elem*sizeof(int));
    hipMalloc(&b_dev, no_elem*sizeof(int));
    hipMalloc(&c_dev, no_elem*sizeof(int));

    hipMemcpy(a_dev,a, no_elem*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(b_dev, b, no_elem*sizeof(int), hipMemcpyHostToDevice);
    GADD<<<1,no_elem>>>(a_dev, b_dev, c_dev);
    hipMemcpy(c, c_dev, no_elem*sizeof(int), hipMemcpyDeviceToHost);
    
    hipFree(a_dev);
    hipFree(b_dev);
    hipFree(c_dev);

}       
